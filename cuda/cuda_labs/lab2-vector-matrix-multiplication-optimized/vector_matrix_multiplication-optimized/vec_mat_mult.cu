#include "hip/hip_runtime.h"
/* Vector-matrix multiplication: Y = A * X.
 * Host code.
 * Author: Naga Kandasamy
 * Date: 11/06/2014
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>

// includes, kernels
#include <vec_mat_mult_kernel.cu>

#define MIN_NUMBER 1
#define MAX_NUMBER 4

////////////////////////////////////////////////////////////////////////////////
// declarations, forward

extern "C"
void compute_gold(float*, const float*, const float*, unsigned int, unsigned int);

Matrix allocate_matrix_on_gpu(const Matrix M);
Matrix allocate_matrix(int num_rows, int num_columns, int init);
void copy_matrix_to_device(Matrix Mdevice, const Matrix Mhost);
void copy_matrix_from_device(Matrix Mhost, const Matrix Mdevice);
void vec_mat_mult_on_device(const Matrix M, const Matrix N, Matrix P);
void print_matrix(const Matrix M);
float get_random_number(int, int);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {

	// Matrices for the program
	Matrix  A; // N x N matrix
	Matrix  X; // N x 1 vector
	Matrix  Y_cpu, Y_gpu; // N x 1 vector
	
	// Initialize the random number generator with a seed value 
	srand(time(NULL));
	
	// Check command line arguments
	if(argc > 1){
		printf("Error. This program accepts no arguments. \n");
		exit(0);
	}		
	 
	// Allocate and initialize the matrices
	A  = allocate_matrix(MATRIX_SIZE, MATRIX_SIZE, 1); // Create a random 512 X 512 matrix
	X  = allocate_matrix(MATRIX_SIZE, 1, 1); // Create a random 512 x 1 vector 
	Y_cpu  = allocate_matrix(MATRIX_SIZE, 1, 0); // Allocate memory for the output vectors
	Y_gpu = allocate_matrix(MATRIX_SIZE, 1, 0); 
 
	
	// Perform the vector-matrix multiplication on the GPU using global memory
	vec_mat_mult_on_device(A, X, Y_gpu);
   
	// compute the vector-matrix multiplication on the CPU for comparison    	
	unsigned int timer;
	cutCreateTimer(&timer);
	cutStartTimer(timer);

	compute_gold(Y_cpu.elements, A.elements, X.elements, A.num_rows, A.num_columns);

	cutStopTimer(timer);
	printf("Execution time on the CPU: %f seconds. \n", (float)cutGetTimerValue(timer)/1000.0);
	
	// check if the device result is equivalent to the expected solution
	int size_elements = NUM_ROWS;
	CUTBoolean res = cutComparefe(Y_cpu.elements, Y_gpu.elements, size_elements, 0.0001f);
	printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");

	// Free host matrices
	free(A.elements); A.elements = NULL;
	free(X.elements); X.elements = NULL;
	free(Y_cpu.elements); Y_cpu.elements = NULL;
	free(Y_gpu.elements); Y_gpu.elements = NULL;


	return 0;
}

// Complete the functionality of vector-matrix multiplication using the GPU 
void vec_mat_mult_on_device(const Matrix A, const Matrix X, Matrix Y){
}



// Allocate a device matrix of same size as M.
Matrix allocate_matrix_on_gpu(const Matrix M){
    Matrix Mdevice = M;
    int size = M.num_rows * M.num_columns * sizeof(float);
    hipMalloc((void**)&Mdevice.elements, size);
    return Mdevice;
}

// Allocate a matrix of dimensions height*width
//	If init == 0, initialize to all zeroes.  
//	If init == 1, perform random initialization.
Matrix allocate_matrix(int num_rows, int num_columns, int init){
    	Matrix M;
    	M.num_columns = M.pitch = num_columns;
    	M.num_rows = num_rows;
    	int size = M.num_rows * M.num_columns;
		
	M.elements = (float*) malloc(size*sizeof(float));
	for(unsigned int i = 0; i < size; i++){
		if(init == 0) M.elements[i] = 0; 
		else
			M.elements[i] = get_random_number(MIN_NUMBER, MAX_NUMBER);
	}
    return M;
}	

// Copy a host matrix to a device matrix.
void copy_matrix_to_device(Matrix Mdevice, const Matrix Mhost)
{
    int size = Mhost.num_rows * Mhost.num_columns * sizeof(float);
    Mdevice.num_rows = Mhost.num_rows;
    Mdevice.num_columns = Mhost.num_columns;
    Mdevice.pitch = Mhost.pitch;
    hipMemcpy(Mdevice.elements, Mhost.elements, size, hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void copy_matrix_from_device(Matrix Mhost, const Matrix Mdevice){
    int size = Mdevice.num_rows * Mdevice.num_columns * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, hipMemcpyDeviceToHost);
}

// Prints the matrix out to screen
void print_matrix(const Matrix M){
	for(unsigned int i = 0; i < M.num_rows; i++){
		for(unsigned int j = 0; j < M.num_columns; j++)
			printf("%f ", M.elements[i*M.num_columns + j]);
		printf("\n");
	} 
	printf("\n");
}

// Returns a random floating-point number between the specified min and max values 
float get_random_number(int min, int max){
	return (float)floor((double)(min + (max - min + 1)*((float)rand()/(float)RAND_MAX)));
}


