#include "hip/hip_runtime.h"
/* Vector-Matrix multiplication: Y = A * X.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "vec_mat_mult.h"

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(float *Ad, float *Xd, float *Yd)
{
	//Multiply A nd X
}



#endif // #ifndef _MATRIXMUL_KERNEL_H_
